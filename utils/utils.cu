#include "hip/hip_runtime.h"
#include <iostream>

#include "utils.hpp"
#include "../main.hpp"

__global__ void convolution_kernel(unsigned char *input_matrix, int *output_matrix, int matrix_width, int matrix_height, float *kernel, int kernel_size) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  int localIdxX = threadIdx.x;
  int localIdxY = threadIdx.y;
  
  int current_matrix_index = globalIdxY*matrix_width + globalIdxX;
  int current_shared_matrix_index = MATRIX_SIZE_PER_BLOCK+2+1+ localIdxY*(MATRIX_SIZE_PER_BLOCK+2) + localIdxX;

  __shared__ unsigned char shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+2)];

  /*
   * x x x x x x MATRIX_SIZE_PER_BLOCK + 2
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x x x x x x
   */
  shared_matrix[current_shared_matrix_index] = input_matrix[current_matrix_index];

  // Handle the borders of each block
  if (localIdxX == 0 && localIdxY == 0) {
    // Fill the edges
    for (int i = 0; i < MATRIX_SIZE_PER_BLOCK+2; i++) {
      // First line
      int first_line_offset = -1;
      if (0 == globalIdxY) {
        first_line_offset = 0;
      }
      shared_matrix[i] = input_matrix[(globalIdxY+first_line_offset)*matrix_width + globalIdxX + i - 1];
      
      // Last line
      int last_line_offset = 0;
      if (globalIdxY+MATRIX_SIZE_PER_BLOCK == matrix_height) {
        last_line_offset = -1;
      }
      shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+1)+i] =
        input_matrix[(globalIdxY+MATRIX_SIZE_PER_BLOCK+last_line_offset)*matrix_width + globalIdxX + i - 1];
    }

    for (int i = 0; i < MATRIX_SIZE_PER_BLOCK; i++) {
      // Left side
      int left_side_offset = -1;
      if (0 == globalIdxX) {
        left_side_offset = 0;
      }
      shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + i*(MATRIX_SIZE_PER_BLOCK+2)] = 
        input_matrix[(globalIdxY+i)*matrix_width + globalIdxX + left_side_offset];

      // Right side
      int right_side_offset = 0;
      if (globalIdxX+MATRIX_SIZE_PER_BLOCK == matrix_width) {
        right_side_offset = -1;
      }
      shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + (i+1)*(MATRIX_SIZE_PER_BLOCK+2) - 1] =
        input_matrix[(globalIdxY+i)*matrix_width + globalIdxX+MATRIX_SIZE_PER_BLOCK + right_side_offset];
    }
  }
  __syncthreads();

  Vec2 index;
  index.x = localIdxX;
  index.y = localIdxY;
  output_matrix[current_matrix_index] = convolution_core(index,
    shared_matrix,
    output_matrix,
    MATRIX_SIZE_PER_BLOCK+2,
    MATRIX_SIZE_PER_BLOCK+2,
    kernel,
    kernel_size);
}

/**
 * Applies discrete convolution over a matrix using a given kernel.
 * This kernel should be called using appropriate number of grids, blocks and threads to match the resolution of the image.
 **/
__device__ __host__ int convolution_core(Vec2 index, unsigned char *input_matrix, int *output_matrix,
  int matrix_width, int matrix_height, float *kernel, int kernel_size
) {
  int convolution_result = 0;

  for (int i = 0; i < kernel_size; i++) {
    for (int j = 0; j < kernel_size; j++) {
      int vertical_offset = ((index.y + i) - (int)floor(kernel_size/2.0));
      int horizontal_offset = (index.x + j) - (int)floor(kernel_size/2.0);
      int tmp_index = vertical_offset*matrix_width + horizontal_offset;
      
      convolution_result += input_matrix[matrix_width+1 + tmp_index] * kernel[i*kernel_size + j];
    }
  }

  if (255 < abs(convolution_result)) {
    convolution_result = convolution_result < 0 ? -255 : 255;
  }
  
  return convolution_result;
}

/**
 * Applies a gaussian blur over a matrix.
 **/
void ProcessingUnitDevice::gaussian_blur(unsigned char *h_matrix, int matrix_width, int matrix_height) {
  const int KERNEL_WIDTH = 3;
  float gaussian_blur_kernel[KERNEL_WIDTH*KERNEL_WIDTH] = {1/16.0, 2/16.0, 1/16.0, 
                                                         2/16.0, 4/16.0, 2/16.0, 
                                                         1/16.0, 2/16.0, 1/16.0};
  int h_int_matrix[matrix_width*matrix_height];
 
  unsigned char *d_input_matrix;
  int *d_output_matrix;
  float *d_kernel;
  hipMalloc((void **) &d_input_matrix, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_output_matrix, matrix_width * matrix_height * sizeof(int));
  hipMalloc((void **) &d_kernel, KERNEL_WIDTH*KERNEL_WIDTH * sizeof(float));

  hipMemcpy(d_input_matrix, h_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, gaussian_blur_kernel, KERNEL_WIDTH*KERNEL_WIDTH * sizeof(int), hipMemcpyHostToDevice);

  dim3 threads = dim3(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 blocks = dim3(matrix_width/MATRIX_SIZE_PER_BLOCK, matrix_height/MATRIX_SIZE_PER_BLOCK);
  std::cout << "Nombre de blocs lancés: " << blocks.x << " " << blocks.y << std::endl;
  convolution_kernel<<<blocks, threads>>>(d_input_matrix, d_output_matrix, matrix_width, matrix_height, d_kernel, KERNEL_WIDTH);
 
  hipMemcpy(h_int_matrix, d_output_matrix, matrix_width*matrix_height*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < matrix_height; i++) {
    for (int j = 0; j < matrix_width; j++) {
      h_matrix[i*matrix_width + j] = h_int_matrix[i*matrix_width + j];
    }
  }

  hipFree(d_input_matrix);
  hipFree(d_output_matrix);
  hipFree(d_kernel);
}

/**
 * Applies a gaussian blur over a matrix.
 **/
void ProcessingUnitHost::gaussian_blur(unsigned char *matrix, int matrix_width, int matrix_height) {
  const int KERNEL_WIDTH = 3;
  float gaussian_blur_kernel[KERNEL_WIDTH*KERNEL_WIDTH] = {1/16.0, 2/16.0, 1/16.0, 
                                                         2/16.0, 4/16.0, 2/16.0, 
                                                         1/16.0, 2/16.0, 1/16.0};
  int int_matrix[matrix_width*matrix_height];
  int *output_matrix = new int[matrix_width * matrix_height];

  for (int i = 0; i < matrix_height; i++) {
    for (int j = 0; j < matrix_width; j++) {
      Vec2 index;
      index.x = j;
      index.y = i;

      output_matrix[i*matrix_width + j] = convolution_core(index, 
        matrix,
        output_matrix,
        matrix_width,
        matrix_height,
        gaussian_blur_kernel,
        KERNEL_WIDTH);
    }
  }
 
  memcpy(int_matrix, output_matrix, matrix_width*matrix_height*sizeof(int));

  for (int i = 0; i < matrix_height; i++) {
    for (int j = 0; j < matrix_width; j++) {
      matrix[i*matrix_width + j] = int_matrix[i*matrix_width + j];
    }
  }

  delete [] output_matrix;
}
