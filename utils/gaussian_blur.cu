#include "hip/hip_runtime.h"

#include <iostream>

#include "gaussian_blur.hpp"
#include "convolution.hpp"
#include "../main.hpp"

/**
 * Applies a gaussian blur over a matrix.
 **/
void ProcessingUnitDevice::gaussian_blur(unsigned char *h_matrix, Dim matrix_dim) {
  const int KERNEL_WIDTH = 3;
  float gaussian_blur_kernel[KERNEL_WIDTH*KERNEL_WIDTH] = {1/16.0, 2/16.0, 1/16.0, 
                                                           2/16.0, 4/16.0, 2/16.0, 
                                                           1/16.0, 2/16.0, 1/16.0};
  int h_int_matrix[matrix_dim.width*matrix_dim.height];
 
  unsigned char *d_input_matrix;
  int *d_output_matrix;
  float *d_kernel;
  hipMalloc(&d_input_matrix, matrix_dim.width * matrix_dim.height * sizeof(unsigned char));
  hipMalloc(&d_output_matrix, matrix_dim.width * matrix_dim.height * sizeof(int));
  hipMalloc(&d_kernel, KERNEL_WIDTH*KERNEL_WIDTH * sizeof(float));

  hipMemcpy(d_input_matrix, h_matrix, matrix_dim.width*matrix_dim.height*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, gaussian_blur_kernel, KERNEL_WIDTH*KERNEL_WIDTH * sizeof(int), hipMemcpyHostToDevice);

  dim3 threads = dim3(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 blocks = dim3(matrix_dim.width/MATRIX_SIZE_PER_BLOCK, matrix_dim.height/MATRIX_SIZE_PER_BLOCK);
  std::cout << "Nombre de blocs lancés: " << blocks.x << " " << blocks.y << std::endl;
  convolution_kernel<<<blocks, threads>>>(d_input_matrix, d_output_matrix, matrix_dim, d_kernel, KERNEL_WIDTH);
 
  hipMemcpy(h_int_matrix, d_output_matrix, matrix_dim.width*matrix_dim.height*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < matrix_dim.height; i++) {
    for (int j = 0; j < matrix_dim.width; j++) {
      h_matrix[i*matrix_dim.width + j] = h_int_matrix[i*matrix_dim.width + j];
    }
  }

  hipFree(d_input_matrix);
  hipFree(d_output_matrix);
  hipFree(d_kernel);
}

/**
 * Applies a gaussian blur over a matrix.
 **/
void ProcessingUnitHost::gaussian_blur(unsigned char *matrix, Dim matrix_dim) {
  const int KERNEL_WIDTH = 3;
  float gaussian_blur_kernel[KERNEL_WIDTH*KERNEL_WIDTH] = {1/16.0, 2/16.0, 1/16.0, 
                                                         2/16.0, 4/16.0, 2/16.0, 
                                                         1/16.0, 2/16.0, 1/16.0};
  int int_matrix[matrix_dim.width*matrix_dim.height];
  int *output_matrix = new int[matrix_dim.width * matrix_dim.height];

  for (int i = 0; i < matrix_dim.height; i++) {
    for (int j = 0; j < matrix_dim.width; j++) {
      int2 index = make_int2(j, i);

      output_matrix[i*matrix_dim.width + j] = convolution_core(index, 
        matrix,
        matrix_dim,
        gaussian_blur_kernel,
        KERNEL_WIDTH);
    }
  }
 
  memcpy(int_matrix, output_matrix, matrix_dim.width*matrix_dim.height*sizeof(int));

  for (int i = 0; i < matrix_dim.height; i++) {
    for (int j = 0; j < matrix_dim.width; j++) {
      matrix[i*matrix_dim.width + j] = int_matrix[i*matrix_dim.width + j];
    }
  }

  delete [] output_matrix;
}
