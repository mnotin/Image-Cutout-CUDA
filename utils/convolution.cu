#include "hip/hip_runtime.h"
#include <iostream>

#include "convolution.hpp"
#include "../main.hpp"

__global__ void convolution_kernel(unsigned char *input_matrix, int *output_matrix, dim3 matrix_dim, const float *kernel, int kernel_size) {
  int2 global_index = make_int2(threadIdx.x + (blockIdx.x * blockDim.x), threadIdx.y + (blockIdx.y * blockDim.y));
  int2 local_index = make_int2(threadIdx.x, threadIdx.y);
  
  int current_mat_idx = global_index.y*matrix_dim.x + global_index.x;
  int current_shared_mat_idx = MATRIX_SIZE_PER_BLOCK+2+1+ local_index.y*(MATRIX_SIZE_PER_BLOCK+2) + local_index.x;

  dim3 shared_matrix_dim(MATRIX_SIZE_PER_BLOCK+2, MATRIX_SIZE_PER_BLOCK+2);
  __shared__ unsigned char shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+2)];

  /*
   * x x x x x x MATRIX_SIZE_PER_BLOCK + 2
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x x x x x x
   */
  shared_matrix[current_shared_mat_idx] = input_matrix[current_mat_idx];

  // Handle borders of the block
  if (local_index.y == 0) {
    // First line
    if (global_index.y == 0) {
      shared_matrix[current_shared_mat_idx - shared_matrix_dim.x] = input_matrix[current_mat_idx];
    } else {
      shared_matrix[current_shared_mat_idx - shared_matrix_dim.x] =
        input_matrix[current_mat_idx - matrix_dim.x];
    }
  } else if (local_index.y == MATRIX_SIZE_PER_BLOCK-1) {
    // Last line
    if (global_index.y == matrix_dim.y-1) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x] = input_matrix[current_mat_idx];
    } else {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x] =
        input_matrix[current_mat_idx + matrix_dim.x];
    }
  }
  
  if (local_index.x == 0) {
    // Left side
    if (global_index.x == 0) {
      shared_matrix[current_shared_mat_idx - 1] = input_matrix[current_mat_idx];
    } else {
      shared_matrix[current_shared_mat_idx - 1] =
        input_matrix[current_mat_idx - 1];
    }
  } else if (local_index.x == MATRIX_SIZE_PER_BLOCK-1) {
    // Right side
    if (global_index.x == matrix_dim.x-1) {
      shared_matrix[current_shared_mat_idx + 1] = input_matrix[current_mat_idx];
    } else {
      shared_matrix[current_shared_mat_idx + 1] =
        input_matrix[current_mat_idx + 1];
    }
  }

  // Handle corners of the block
  if (local_index.x == 0 && local_index.y == 0) {
    // Top left
    if (global_index.x == 0 && global_index.y == 0) {
      shared_matrix[0] = input_matrix[current_mat_idx];
    } else if (global_index.x == 0) {
      shared_matrix[0] = input_matrix[current_mat_idx - matrix_dim.x];
    } else if (global_index.y == 0) {
      shared_matrix[0] = input_matrix[current_mat_idx - 1];
    } else {
      shared_matrix[0] = input_matrix[current_mat_idx - matrix_dim.x - 1];
    }
  } else if (local_index.x == MATRIX_SIZE_PER_BLOCK-1 && local_index.y == 0) {
    // Top right
    if (global_index.x == matrix_dim.x-1 && global_index.y == 0) {
      shared_matrix[MATRIX_SIZE_PER_BLOCK+1] = input_matrix[current_mat_idx];
    } else if (global_index.x == matrix_dim.x-1) {
      shared_matrix[MATRIX_SIZE_PER_BLOCK+1] = input_matrix[current_mat_idx - matrix_dim.x];
    } else if (global_index.y == 0) {
      shared_matrix[MATRIX_SIZE_PER_BLOCK+1] = input_matrix[current_mat_idx + 1];
    } else {
      shared_matrix[MATRIX_SIZE_PER_BLOCK+1] = input_matrix[current_mat_idx - matrix_dim.x + 1];
    }
  } else if (local_index.x == 0 && local_index.y == MATRIX_SIZE_PER_BLOCK-1) {
    // Bottom left
    if (global_index.x == 0 && global_index.y == matrix_dim.y-1) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x - 1] = input_matrix[current_mat_idx];
    } else if (global_index.x == 0) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x - 1] = input_matrix[current_mat_idx + matrix_dim.x];
    } else if (global_index.y == matrix_dim.y-1) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x - 1] = input_matrix[current_mat_idx - 1];
    } else {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x - 1] = input_matrix[current_mat_idx + matrix_dim.x - 1];
    }
  } else if (local_index.x == MATRIX_SIZE_PER_BLOCK-1 && local_index.y == MATRIX_SIZE_PER_BLOCK-1) {
    // Bottom right
    if (global_index.x == matrix_dim.x-1 && global_index.y == matrix_dim.y-1) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x + 1] = input_matrix[current_mat_idx];
    } else if (global_index.x == matrix_dim.x-1) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x + 1] = input_matrix[current_mat_idx + matrix_dim.x];
    } else if (global_index.y == matrix_dim.y-1) {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x + 1] = input_matrix[current_mat_idx + 1];
    } else {
      shared_matrix[current_shared_mat_idx + shared_matrix_dim.x + 1] = input_matrix[current_mat_idx + matrix_dim.x + 1];
    }
  }

  __syncthreads();

  output_matrix[current_mat_idx] = convolution_core(
    local_index,
    shared_matrix,
    shared_matrix_dim,
    kernel,
    kernel_size);
}

/**
 * Applies discrete convolution over a matrix using a given kernel.
 * This kernel should be called using appropriate number of grids, blocks and threads to match the resolution of the image.
 **/
__device__ __host__ int convolution_core(int2 index, unsigned char *input_matrix, dim3 matrix_dim, const float *kernel, int kernel_size) {
  int convolution_result = 0;

  for (int i = 0; i < kernel_size; i++) {
    for (int j = 0; j < kernel_size; j++) {
      int vertical_offset = index.y - floor(kernel_size / (float) 2.0) + i;
      int horizontal_offset = index.x - floor(kernel_size / (float) 2.0) + j;
      int tmp_index = vertical_offset*matrix_dim.x + horizontal_offset;
      
      convolution_result += input_matrix[matrix_dim.x +1 + tmp_index] * kernel[i*kernel_size + j];
    }
  }

  if (255 < abs(convolution_result)) {
    convolution_result = convolution_result < 0 ? -255 : 255;
  }
  
  return convolution_result;
}
