#include "hip/hip_runtime.h"
#include <iostream>

#include "convolution.hpp"
#include "../main.hpp"

__global__ void convolution_kernel(unsigned char *input_matrix, int *output_matrix, Dim matrix_dim, float *kernel, int kernel_size) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  int localIdxX = threadIdx.x;
  int localIdxY = threadIdx.y;
  
  int current_matrix_index = globalIdxY*matrix_dim.width + globalIdxX;
  int current_shared_matrix_index = MATRIX_SIZE_PER_BLOCK+2+1+ localIdxY*(MATRIX_SIZE_PER_BLOCK+2) + localIdxX;

  __shared__ unsigned char shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+2)];

  /*
   * x x x x x x MATRIX_SIZE_PER_BLOCK + 2
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x x x x x x
   */
  shared_matrix[current_shared_matrix_index] = input_matrix[current_matrix_index];

  // Handle the borders of each block
  if (localIdxX == 0 && localIdxY == 0) {
    // Fill the edges
    for (int i = 0; i < MATRIX_SIZE_PER_BLOCK+2; i++) {
      // First line
      int first_line_offset = -1;
      if (0 == globalIdxY) {
        first_line_offset = 0;
      }
      shared_matrix[i] = input_matrix[(globalIdxY+first_line_offset)*matrix_dim.width + globalIdxX + i - 1];
      
      // Last line
      int last_line_offset = 0;
      if (globalIdxY+MATRIX_SIZE_PER_BLOCK == matrix_dim.height) {
        last_line_offset = -1;
      }
      shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+1)+i] =
        input_matrix[(globalIdxY+MATRIX_SIZE_PER_BLOCK+last_line_offset)*matrix_dim.width + globalIdxX + i - 1];
    }

    for (int i = 0; i < MATRIX_SIZE_PER_BLOCK; i++) {
      // Left side
      int left_side_offset = -1;
      if (0 == globalIdxX) {
        left_side_offset = 0;
      }
      shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + i*(MATRIX_SIZE_PER_BLOCK+2)] = 
        input_matrix[(globalIdxY+i)*matrix_dim.width + globalIdxX + left_side_offset];

      // Right side
      int right_side_offset = 0;
      if (globalIdxX+MATRIX_SIZE_PER_BLOCK == matrix_dim.width) {
        right_side_offset = -1;
      }
      shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + (i+1)*(MATRIX_SIZE_PER_BLOCK+2) - 1] =
        input_matrix[(globalIdxY+i)*matrix_dim.width + globalIdxX+MATRIX_SIZE_PER_BLOCK + right_side_offset];
    }
  }
  __syncthreads();

  Vec2 index;
  index.x = localIdxX;
  index.y = localIdxY;
  Dim shared_matrix_dim;
  shared_matrix_dim.width = MATRIX_SIZE_PER_BLOCK+2;
  shared_matrix_dim.height = MATRIX_SIZE_PER_BLOCK+2;
  output_matrix[current_matrix_index] = convolution_core(index,
    shared_matrix,
    output_matrix,
    shared_matrix_dim,
    kernel,
    kernel_size);
}

/**
 * Applies discrete convolution over a matrix using a given kernel.
 * This kernel should be called using appropriate number of grids, blocks and threads to match the resolution of the image.
 **/
__device__ __host__ int convolution_core(Vec2 index, unsigned char *input_matrix, int *output_matrix,
  Dim matrix_dim, float *kernel, int kernel_size
) {
  int convolution_result = 0;

  for (int i = 0; i < kernel_size; i++) {
    for (int j = 0; j < kernel_size; j++) {
      int vertical_offset = ((index.y + i) - (int)floor(kernel_size/2.0));
      int horizontal_offset = (index.x + j) - (int)floor(kernel_size/2.0);
      int tmp_index = vertical_offset*matrix_dim.width + horizontal_offset;
      
      convolution_result += input_matrix[matrix_dim.width+1 + tmp_index] * kernel[i*kernel_size + j];
    }
  }

  if (255 < abs(convolution_result)) {
    convolution_result = convolution_result < 0 ? -255 : 255;
  }
  
  return convolution_result;
}
