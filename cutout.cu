#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "cutout.h"
#include "main.h"
#include "utils.h"

void cutout(unsigned char *h_rgb_image, unsigned char *h_edge_matrix, int matrix_width, int matrix_height, int start_pixel_x, int start_pixel_y, int threshold) {
  int h_done = 0;
  unsigned char h_cutout_matrix[matrix_height][matrix_width];
 
  for (int i = 0; i < matrix_height; i++) {
    for (int j = 0; j < matrix_width; j++) {
      h_cutout_matrix[i][j] = 'D';
    }
  }

  unsigned char *d_rgb_image;
  unsigned char *d_edge_matrix;
  unsigned char *d_cutout_matrix;
  int *d_done;

  hipMalloc((void **) &d_rgb_image, 3 * matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_edge_matrix, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_cutout_matrix, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_done, sizeof(int));

  hipMemcpy(d_rgb_image, h_rgb_image, 3 * matrix_width * matrix_height * sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_edge_matrix, h_edge_matrix, matrix_width * matrix_height * sizeof(unsigned char), hipMemcpyHostToDevice);
  for (int i = 0; i < matrix_height; i++) {
    hipMemcpy(d_cutout_matrix+i*matrix_width, h_cutout_matrix[i], matrix_width * sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  
  dim3 threads = dim3(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 blocks = dim3(matrix_width/MATRIX_SIZE_PER_BLOCK, matrix_height/MATRIX_SIZE_PER_BLOCK);
  draw_edges_on_cutout_matrix<<<blocks, threads>>>(d_edge_matrix, d_cutout_matrix, matrix_width, matrix_height, start_pixel_x, start_pixel_y, threshold);

  while (h_done == 0) {
    h_done = 1;
    hipMemcpy(d_done, &h_done, sizeof(int), hipMemcpyHostToDevice);
    cutout_algorithm<<<blocks, threads>>>(d_cutout_matrix, matrix_width, matrix_height, d_done);
    hipMemcpy(&h_done, d_done, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  apply_cutout<<<blocks, threads>>>(d_cutout_matrix, d_rgb_image, matrix_width, matrix_height, start_pixel_x, start_pixel_y);

  hipMemcpy(h_rgb_image, d_rgb_image, 3 * matrix_width * matrix_height * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(d_rgb_image);
  hipFree(d_edge_matrix);
  hipFree(d_cutout_matrix);
  hipFree(d_done);
}

/**
 * First step of the cutout process.
 * Each gradient pixel with a value above the threshold is considered a border.
 **/
__global__ void draw_edges_on_cutout_matrix(unsigned char *edge_matrix, unsigned char *cutout_matrix, int matrix_width, int matrix_height, int start_pixel_x, int start_pixel_y, int threshold) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  const int GLOBAL_IDX = globalIdxY * matrix_width + globalIdxX;

  if (globalIdxX < matrix_width && globalIdxY < matrix_height && threshold < edge_matrix[GLOBAL_IDX]) {
    cutout_matrix[GLOBAL_IDX] = 'B'; 
  }
  
  if (start_pixel_x == globalIdxX && start_pixel_y == globalIdxY) {
    cutout_matrix[start_pixel_y*matrix_width + start_pixel_x] = 'A';
  }
}

/**
 * Main part of the cutout process.
 * Loops over a cutout matrix from the start pixel to fill the shape it is in.
 **/
__global__ void cutout_algorithm(unsigned char *cutout_matrix, int matrix_width, int matrix_height, int *done) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  int localIdxX = threadIdx.x;
  int localIdxY = threadIdx.y;

  __shared__ int shared_done;

  if (localIdxX == 0 && localIdxY == 0) {
    shared_done = 1; // Initialize the variable of the block
  }
  
  __syncthreads();

  // Process
  if (cutout_matrix[globalIdxY*matrix_width + globalIdxX] == 'A') {
    // Active pixel
    if (cutout_matrix[globalIdxY*matrix_width + globalIdxX-1] == 'D') {
      cutout_matrix[globalIdxY*matrix_width + globalIdxX-1] = 'A';
      shared_done = 0;
    }
    
    if (cutout_matrix[globalIdxY*matrix_width + globalIdxX+1] == 'D') {
      cutout_matrix[globalIdxY*matrix_width + globalIdxX+1] = 'A';
      shared_done = 0;
    }
    
    if (cutout_matrix[(globalIdxY-1)*matrix_width + globalIdxX] == 'D') {
      cutout_matrix[(globalIdxY-1)*matrix_width + globalIdxX] = 'A';
      shared_done = 0;
    }
    
    if (cutout_matrix[(globalIdxY+1)*matrix_width + globalIdxX] == 'D') {
      cutout_matrix[(globalIdxY+1)*matrix_width + globalIdxX] = 'A';
      shared_done = 0;
    }
    
    cutout_matrix[globalIdxY*matrix_width + globalIdxX] = 'M';
  }
 
  if (localIdxX == 0 && localIdxY == 0 && shared_done == 0) {
    *done = 0;
  }
}

__global__ void apply_cutout(unsigned char *cutout_matrix, unsigned char *output_image, int image_width, int image_height, int start_pixel_x, int start_pixel_y) { 
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  const int GLOBAL_IDX = globalIdxY * image_width + globalIdxX;
 
  if (globalIdxX == start_pixel_x && globalIdxY == start_pixel_y) {
    output_image[3 * (GLOBAL_IDX)] = 255;
    output_image[3 * (GLOBAL_IDX) + 1] = 0; 
    output_image[3 * (GLOBAL_IDX) + 2] = 0; 
  } else if (cutout_matrix[globalIdxY*image_width + globalIdxX] != 'M') {
    output_image[3 * (GLOBAL_IDX)] = 0; 
    output_image[3 * (GLOBAL_IDX) + 1] = 0; 
    output_image[3 * (GLOBAL_IDX) + 2] = 0; 
  }
}

