#include "hip/hip_runtime.h"
#include <math.h>
#include <unistd.h>

#include "cutout.hpp"
#include "main.hpp"

void ProcessingUnitDevice::cutout(unsigned char *h_rgb_image, unsigned char *h_edge_matrix, Dim matrix_dim, Vec2 start_pixel, int threshold) {
  int h_done = 0;
  unsigned char h_cutout_matrix[matrix_dim.height][matrix_dim.width];
  
  for (int i = 0; i < matrix_dim.height; i++) {
    for (int j = 0; j < matrix_dim.width; j++) {
      h_cutout_matrix[i][j] = 'D';
    }
  }

  unsigned char *d_rgb_image;
  unsigned char *d_edge_matrix;
  unsigned char *d_cutout_matrix;
  int *d_done;

  hipMalloc((void **) &d_rgb_image, 3 * matrix_dim.width * matrix_dim.height * sizeof(unsigned char));
  hipMalloc((void **) &d_edge_matrix, matrix_dim.width * matrix_dim.height * sizeof(unsigned char));
  hipMalloc((void **) &d_cutout_matrix, matrix_dim.width * matrix_dim.height * sizeof(unsigned char));
  hipMalloc((void **) &d_done, sizeof(int));

  hipMemcpy(d_rgb_image, h_rgb_image, 3 * matrix_dim.width * matrix_dim.height * sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_edge_matrix, h_edge_matrix, matrix_dim.width * matrix_dim.height * sizeof(unsigned char), hipMemcpyHostToDevice);
  for (int i = 0; i < matrix_dim.height; i++) {
    hipMemcpy(d_cutout_matrix+i*matrix_dim.width, h_cutout_matrix[i], matrix_dim.width * sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  
  dim3 threads = dim3(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 blocks = dim3(matrix_dim.width/MATRIX_SIZE_PER_BLOCK, matrix_dim.height/MATRIX_SIZE_PER_BLOCK);
  draw_edges_on_cutout_matrix_kernel<<<blocks, threads>>>(d_edge_matrix, d_cutout_matrix, matrix_dim, start_pixel, threshold);

  while (h_done == 0) {
    h_done = 1;
    hipMemcpy(d_done, &h_done, sizeof(int), hipMemcpyHostToDevice);
    cutout_algorithm_kernel<<<blocks, threads>>>(d_cutout_matrix, matrix_dim, d_done);
    hipMemcpy(&h_done, d_done, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  apply_cutout_kernel<<<blocks, threads>>>(d_cutout_matrix, d_rgb_image, matrix_dim, start_pixel);

  hipMemcpy(h_rgb_image, d_rgb_image, 3 * matrix_dim.width * matrix_dim.height * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(d_rgb_image);
  hipFree(d_edge_matrix);
  hipFree(d_cutout_matrix);
  hipFree(d_done);
}

void ProcessingUnitHost::cutout(unsigned char *rgb_image, unsigned char *edge_matrix, Dim matrix_dim, Vec2 start_pixel, int threshold) {
  int done = 0;
  unsigned char cutout_matrix[matrix_dim.height * matrix_dim.width];
  
  for (int i = 0; i < matrix_dim.height; i++) {
    for (int j = 0; j < matrix_dim.width; j++) {
      cutout_matrix[i*matrix_dim.width + j] = 'D';
    }
  }
  
  Vec2 index;
  for (index.y = 0; index.y < matrix_dim.height; index.y++) {
    for (index.x = 0; index.x < matrix_dim.width; index.x++) {
      cutout_matrix[index.y*matrix_dim.width + index.x] = draw_edges_on_cutout_matrix_core(index, edge_matrix, matrix_dim, start_pixel, threshold);
    }
  }

  while (done == 0) {
    done = 1;
    for (index.y = 0; index.y < matrix_dim.width; index.y++) {
      for (index.x = 0; index.x < matrix_dim.height; index.x++) {
        cutout_algorithm_core(index, cutout_matrix, matrix_dim, &done);
      }
    }
  }
  
  for (index.y = 0; index.y < matrix_dim.height; index.y++) {
    for (index.x = 0; index.x < matrix_dim.width; index.x++) {
      apply_cutout_core(index, cutout_matrix, rgb_image, matrix_dim, start_pixel);
    }
  }
}

/**
 * First step of the cutout process.
 * Each gradient pixel with a value above the threshold is considered a border.
 **/
__global__ void draw_edges_on_cutout_matrix_kernel(unsigned char *edge_matrix, unsigned char *cutout_matrix, Dim matrix_dim, Vec2 start_pixel, int threshold) {
  Vec2 index;
  index.x = threadIdx.x + (blockIdx.x * blockDim.x);
  index.y = threadIdx.y + (blockIdx.y * blockDim.y);
  
  cutout_matrix[index.y*matrix_dim.width + index.x] = draw_edges_on_cutout_matrix_core(index, edge_matrix, matrix_dim, start_pixel, threshold);
}

__device__ __host__ unsigned char draw_edges_on_cutout_matrix_core(Vec2 index, unsigned char *edge_matrix, Dim matrix_dim, Vec2 start_pixel, int threshold) {
  unsigned char result = 'D';

  if (index.x < matrix_dim.width && index.y < matrix_dim.height && threshold < edge_matrix[index.y*matrix_dim.width + index.x]) {
    result = 'B'; 
  }
  
  if (start_pixel.x == index.x && start_pixel.y == index.y) {
    result = 'A';
  }

  return result;
}

/**
 * Main part of the cutout process.
 * Loops over a cutout matrix from the start pixel to fill the shape it is in.
 **/
__global__ void cutout_algorithm_kernel(unsigned char *cutout_matrix, Dim matrix_dim, int *done) {
  Vec2 global_index;
  Vec2 local_index;
  global_index.x = threadIdx.x + (blockIdx.x * blockDim.x);
  global_index.y = threadIdx.y + (blockIdx.y * blockDim.y);
  local_index.x = threadIdx.x;
  local_index.y = threadIdx.y;

  __shared__ int shared_done;

  if (local_index.x == 0 && local_index.y == 0) {
    shared_done = 1; // Initialize the variable of the block
  }
  
  __syncthreads();

  cutout_algorithm_core(global_index, cutout_matrix, matrix_dim, &shared_done);

  // The first local thread has to wait for all the threads of the bloc to finish
  __syncthreads();
 
  if (local_index.x == 0 && local_index.y == 0 && shared_done == 0) {
    *done = 0;
  }
}

__device__ __host__ void cutout_algorithm_core(Vec2 index, unsigned char *cutout_matrix, Dim matrix_dim, int *done) {
  const int INT_INDEX = index.y*matrix_dim.width + index.x;
  if (cutout_matrix[INT_INDEX] == 'A') {
    // Active pixel
    if (0 < index.x && cutout_matrix[INT_INDEX-1] == 'D') {
      cutout_matrix[INT_INDEX-1] = 'A';
      *done = 0;
    }
    
    if (index.x < matrix_dim.width-1 && cutout_matrix[INT_INDEX+1] == 'D') {
      cutout_matrix[INT_INDEX+1] = 'A';
      *done = 0;
    }
    
    if (0 < index.y && cutout_matrix[INT_INDEX - matrix_dim.width] == 'D') {
      cutout_matrix[INT_INDEX - matrix_dim.width] = 'A';
      *done = 0;
    }
    
    if (index.y < matrix_dim.height-1 && cutout_matrix[INT_INDEX + matrix_dim.width] == 'D') {
      cutout_matrix[INT_INDEX + matrix_dim.width] = 'A';
      *done = 0;
    }
      
    cutout_matrix[INT_INDEX] = 'M'; // At the end of the loop, current pixel is marked
  }
}

__global__ void apply_cutout_kernel(unsigned char *cutout_matrix, unsigned char *output_image, Dim image_dim, Vec2 start_pixel) { 
  Vec2 index;
  index.x = threadIdx.x + (blockIdx.x * blockDim.x);
  index.y = threadIdx.y + (blockIdx.y * blockDim.y);
  
  apply_cutout_core(index, cutout_matrix, output_image, image_dim, start_pixel);
}

__device__ __host__ void apply_cutout_core(Vec2 index, unsigned char *cutout_matrix, unsigned char *output_image, Dim image_dim, Vec2 start_pixel) { 
  const int INT_INDEX = index.y*image_dim.width + index.x;

  if (index.x == start_pixel.x && index.y == start_pixel.y) {
    output_image[3 * (INT_INDEX)] = 255;
    output_image[3 * (INT_INDEX) + 1] = 0; 
    output_image[3 * (INT_INDEX) + 2] = 0; 
  } else if (cutout_matrix[INT_INDEX] == 'M') {
    output_image[3 * (INT_INDEX)] = 0; 
    output_image[3 * (INT_INDEX) + 1] = 0; 
    output_image[3 * (INT_INDEX) + 2] = 0; 
  }
}
