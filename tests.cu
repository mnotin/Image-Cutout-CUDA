#include <iostream>

#include "tests.hpp"
#include "utils/convolution.hpp"
#include "utils/rgb_to_gray.hpp"
#include "utils/gaussian_blur.hpp"
#include "cutout.hpp"

#include "img.h"
#include <opencv2/opencv.hpp>

#include "edge_detection/sobel_feldman.hpp"
#include "edge_detection/canny.hpp"

void test_sobel_feldman(char *filename, int start_pixel_x, int start_pixel_y, ProcessingUnit processing_unit) {
  RGBImage *rgb_image = readPPM(filename);
  GrayImage *gray_image = createPGM(rgb_image->width, rgb_image->height);
  GrayImage *gradient_image = createPGM(rgb_image->width, rgb_image->height);
  float *angle_image = new float[rgb_image->width * rgb_image->height];
  RGBImage *edge_color_image = readPPM(filename);

  if (rgb_image == NULL) {
    std::cout << "Error reading the image" << std::endl;
    exit(EXIT_FAILURE);
  }

  if (processing_unit == ProcessingUnit::Device) {
    // GPU
    // 1. First step, convert the picture into grayscale
    ProcessingUnitDevice::rgb_to_gray(rgb_image, gray_image);

    // 2. Second step, smooth the image using a Gaussian blur
    // to remove possible noise in the picture
    for (int i = 0; i < 5; i++) {
      ProcessingUnitDevice::gaussian_blur(gray_image->data, gray_image->width, gray_image->height);
      hipDeviceSynchronize();
    }

    // 3. Third step, apply the Sobel-Feldman operator to detect edges of shapes
    sobel_feldman(gray_image->data, gradient_image->data, angle_image, gray_image->width, gray_image->height);
    writePGM("output/sf_gradient_output.pgm", gradient_image);

    generate_edge_color(gradient_image->data, angle_image, edge_color_image->data, edge_color_image->width, edge_color_image->height);
    writePPM("output/edge_color_output.ppm", edge_color_image);

    // 4. Last step, cutout the object selected by the user
    cutout(rgb_image->data, gradient_image->data, gray_image->width, gray_image->height, start_pixel_x, start_pixel_y, 0);
  } else if (processing_unit == ProcessingUnit::Host) {
    // CPU
  }
  
  writePPM("output/cutout_output.ppm", rgb_image);

  destroyPPM(rgb_image);
  destroyPGM(gray_image);  
  destroyPGM(gradient_image);  
  destroyPPM(edge_color_image);  
  delete [] angle_image;
}

void test_canny(char *filename, int start_pixel_x, int start_pixel_y, int canny_min,
  int canny_max, int canny_sample_offset, ProcessingUnit processing_unit
) {
  RGBImage *rgb_image = readPPM(filename);
  GrayImage *gray_image = createPGM(rgb_image->width, rgb_image->height);
  GrayImage *gradient_image = createPGM(rgb_image->width, rgb_image->height);
  float *angle_image = new float[rgb_image->width * rgb_image->height];
  RGBImage *edge_color_image = readPPM(filename);
   if (rgb_image == NULL) {
    std::cout << "Error reading the image" << std::endl;
    exit(EXIT_FAILURE);
  }
 
  if (processing_unit == ProcessingUnit::Device) {
    // GPU
    // 1. First step, convert the picture into grayscale
    ProcessingUnitDevice::rgb_to_gray(rgb_image, gray_image);


    // 2. Second step, smooth the image using a Gaussian blur
    // to remove possible noise in the picture
    for (int i = 0; i < 5; i++) {
      ProcessingUnitDevice::gaussian_blur(gray_image->data, gray_image->width, gray_image->height);
      hipDeviceSynchronize();
    }

    // 3. Third step, apply the Sobel-Feldman operator to detect edges of shapes
    sobel_feldman(gray_image->data, gradient_image->data, angle_image, gray_image->width, gray_image->height);
    writePGM("output/sf_gradient_output.pgm", gradient_image);

    generate_edge_color(gradient_image->data, angle_image, edge_color_image->data, edge_color_image->width, edge_color_image->height);
    writePPM("output/edge_color_output.ppm", edge_color_image);

    GrayImage *buffer_gray = createPGM(gradient_image->width, gradient_image->height);
    RGBImage *buffer_rgb = createPPM(gradient_image->width, gradient_image->height);
    int file_index = 0;
    for (int i = canny_min; i <= canny_max && canny_sample_offset; i += canny_sample_offset) {
      memcpy(buffer_gray->data, gradient_image->data, sizeof(unsigned char) * gradient_image->width * gradient_image->height);
      canny(buffer_gray->data, angle_image, buffer_gray->width, buffer_gray->height, i, canny_max);

      // Create the name of the output file
      const char *prefix_gray = "output/canny_output";
      char number_gray[4] = "000";
      sprintf(number_gray, "%d", file_index);
      char filename_gray[strlen(prefix_gray) + 3 + 4 + 1]; // prefix + number + .ppm + \0
      bzero(filename_gray, strlen(prefix_gray) + 3 + 4 + 1);
      strcpy(filename_gray, prefix_gray);
      strcpy(filename_gray + strlen(prefix_gray), number_gray);
      strcpy(filename_gray + strlen(filename_gray), ".ppm");

      //printf("%s\n", filename_gray);
      //writePGM(filename, buffer);
    
      // 4. Last step, cutout the object selected by the user
      memcpy(buffer_rgb->data, rgb_image->data, sizeof(unsigned char) * gradient_image->width * gradient_image->height * 3);
      cutout(buffer_rgb->data, buffer_gray->data, gray_image->width, gray_image->height, start_pixel_x, start_pixel_y, 0);

      const char *prefix_rgb = "output/cutout_output";
      char number_rgb[4] = "000";
      sprintf(number_rgb, "%d", file_index);
      char filename_rgb[strlen(prefix_rgb) + 3 + 4 + 1]; // prefix + number + .ppm + \0
      bzero(filename_rgb, strlen(prefix_rgb) + 3 + 4 + 1);
      strcpy(filename_rgb, prefix_rgb);
      strcpy(filename_rgb + strlen(prefix_rgb), number_rgb);
      strcpy(filename_rgb + strlen(filename_rgb), ".ppm");
      printf("%s\n", filename_rgb);
      writePPM(filename_rgb, buffer_rgb);
  
      file_index += 1;
    }
    destroyPGM(buffer_gray);
    destroyPPM(buffer_rgb);
  } else if (processing_unit == ProcessingUnit::Host) {
  }

  destroyPPM(rgb_image);
  destroyPGM(gray_image);  
  destroyPGM(gradient_image);  
  destroyPPM(edge_color_image);  
  delete [] angle_image; 
}
