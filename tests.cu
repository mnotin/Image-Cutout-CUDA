#include <stdio.h>

#include "tests.h"
#include "main.h"
#include "edge_detection.h"
#include "img.h"

void test_sobel_feldman(char *filename) {
  GrayImage* grayImage = readPGM(filename);

  if (grayImage == NULL) {
    printf("Error reading the image\n");
    exit(EXIT_FAILURE);
  } 

  for (int i = 0; i < 10; i++) {
    gaussian_blur(grayImage->data, grayImage->width, grayImage->height);
    hipDeviceSynchronize();
  }
  sobel_feldman(grayImage->data, grayImage->width, grayImage->height);
  
  writePGM("sobel_feldman_output.pgm", grayImage);
  destroyPGM(grayImage);  
}
