#include <iostream>

#include "tests.hpp"
#include "utils/convolution.hpp"
#include "utils/rgb_to_gray.hpp"
#include "utils/gaussian_blur.hpp"
#include "cutout.hpp"

#include "img.h"

#include "edge_detection/sobel_feldman.hpp"
#include "edge_detection/canny.hpp"

void test_sobel_feldman(char *filename, Vec2 start_pixel, ProcessingUnit processing_unit) {
  RGBImage *rgb_image = readPPM(filename);
  GrayImage *gray_image = createPGM(rgb_image->width, rgb_image->height);
  GrayImage *gradient_image = createPGM(rgb_image->width, rgb_image->height);
  float *angle_image = new float[rgb_image->width * rgb_image->height];
  RGBImage *edge_color_image = readPPM(filename);

  Dim rgb_image_dim;
  rgb_image_dim.width = rgb_image->width;
  rgb_image_dim.height = rgb_image->height;
  Dim gray_image_dim;
  gray_image_dim.width = gray_image->width;
  gray_image_dim.height = gray_image->height;

  if (rgb_image == NULL) {
    std::cout << "Error reading the image" << std::endl;
    exit(EXIT_FAILURE);
  }

  if (processing_unit == ProcessingUnit::Device) {
    // GPU
    // 1. First step, convert the picture into grayscale
    ProcessingUnitDevice::rgb_to_gray(rgb_image, gray_image);

    // 2. Second step, smooth the image using a Gaussian blur
    // to remove possible noise in the picture
    for (int i = 0; i < 5; i++) {
      ProcessingUnitDevice::gaussian_blur(gray_image->data, gray_image_dim);
      hipDeviceSynchronize();
    }

    // 3. Third step, apply the Sobel-Feldman operator to detect edges of shapes
    ProcessingUnitDevice::sobel_feldman(gray_image->data, gradient_image->data, angle_image, gray_image_dim);
    writePGM("output/sf_gradient_output.pgm", gradient_image);

    ProcessingUnitDevice::generate_edge_color(gradient_image->data, angle_image, edge_color_image->data, rgb_image_dim);
    writePPM("output/edge_color_output.ppm", edge_color_image);

    // 4. Last step, cutout the object selected by the user
    ProcessingUnitDevice::cutout(rgb_image->data, gradient_image->data, gray_image_dim, start_pixel, 0);
  } else if (processing_unit == ProcessingUnit::Host) {
    // CPU
    // 1. First step, convert the picture into grayscale
    ProcessingUnitHost::rgb_to_gray(rgb_image, gray_image);
    
    // 2. Second step, smooth the image using a Gaussian blur
    // to remove possible noise in the picture
    for (int i = 0; i < 5; i++) {
      ProcessingUnitHost::gaussian_blur(gray_image->data, gray_image_dim);
    }
    
    // 3. Third step, apply the Sobel-Feldman operator to detect edges of shapes
    ProcessingUnitHost::sobel_feldman(gray_image->data, gradient_image->data, angle_image, gray_image_dim);
    writePGM("output/sf_gradient_output.pgm", gradient_image);
    
    ProcessingUnitHost::generate_edge_color(gradient_image->data, angle_image, edge_color_image->data, rgb_image_dim);
    writePPM("output/edge_color_output.ppm", edge_color_image);
    
    // 4. Last step, cutout the object selected by the user
    ProcessingUnitHost::cutout(rgb_image->data, gradient_image->data, gray_image_dim, start_pixel, 0);
  }
  
  writePPM("output/cutout_output.ppm", rgb_image);

  destroyPPM(rgb_image);
  destroyPGM(gray_image);  
  destroyPGM(gradient_image);  
  destroyPPM(edge_color_image);  
  delete [] angle_image;
}

void test_canny(char *filename, Vec2 start_pixel, int canny_min,
  int canny_max, int canny_sample_offset, ProcessingUnit processing_unit
) {
  RGBImage *rgb_image = readPPM(filename);
  GrayImage *gray_image = createPGM(rgb_image->width, rgb_image->height);
  GrayImage *gradient_image = createPGM(rgb_image->width, rgb_image->height);
  float *angle_image = new float[rgb_image->width * rgb_image->height];
  RGBImage *edge_color_image = readPPM(filename);

  Dim rgb_image_dim;
  rgb_image_dim.width = rgb_image->width;
  rgb_image_dim.height = rgb_image->height;
  Dim gray_image_dim;
  gray_image_dim.width = gray_image->width;
  gray_image_dim.height = gray_image->height;
  
  if (rgb_image == NULL) {
    std::cout << "Error reading the image" << std::endl;
    exit(EXIT_FAILURE);
  }
 
  if (processing_unit == ProcessingUnit::Device) {
    // GPU
    // 1. First step, convert the picture into grayscale
    ProcessingUnitDevice::rgb_to_gray(rgb_image, gray_image);
    
    // 2. Second step, smooth the image using a Gaussian blur
    // to remove possible noise in the picture
    for (int i = 0; i < 5; i++) {
      ProcessingUnitDevice::gaussian_blur(gray_image->data, gray_image_dim);
      hipDeviceSynchronize();
    }
    writePGM("output/blurred_image_output.pgm", gray_image);
    
    // 3. Third step, apply the Sobel-Feldman operator to detect edges of shapes
    ProcessingUnitDevice::sobel_feldman(gray_image->data, gradient_image->data, angle_image, gray_image_dim);
    writePGM("output/sf_gradient_output.pgm", gradient_image);

    ProcessingUnitDevice::generate_edge_color(gradient_image->data, angle_image, edge_color_image->data, rgb_image_dim);
    writePPM("output/edge_color_output.ppm", edge_color_image);

    GrayImage *buffer_gray = createPGM(gradient_image->width, gradient_image->height);
    RGBImage *buffer_rgb = createPPM(gradient_image->width, gradient_image->height);
    int file_index = 0;
    for (int i = canny_min; i <= canny_max && canny_sample_offset; i += canny_sample_offset) {
      memcpy(buffer_gray->data, gradient_image->data, sizeof(unsigned char) * gradient_image->width * gradient_image->height);
      ProcessingUnitDevice::canny(buffer_gray->data, angle_image, gray_image_dim, i, canny_max);

      // Create the name of the output file
      const char *prefix_gray = "output/canny_output";
      char number_gray[4] = "000";
      sprintf(number_gray, "%d", file_index);
      char filename_gray[strlen(prefix_gray) + 3 + 4 + 1]; // prefix + number + .ppm + \0
      bzero(filename_gray, strlen(prefix_gray) + 3 + 4 + 1);
      strcpy(filename_gray, prefix_gray);
      strcpy(filename_gray + strlen(prefix_gray), number_gray);
      strcpy(filename_gray + strlen(filename_gray), ".ppm");

      //printf("%s\n", filename_gray);
      //writePGM(filename, buffer);
    
      // 4. Last step, cutout the object selected by the user
      memcpy(buffer_rgb->data, rgb_image->data, sizeof(unsigned char) * gradient_image->width * gradient_image->height * 3);
      ProcessingUnitDevice::cutout(buffer_rgb->data, buffer_gray->data, gray_image_dim, start_pixel, 0);

      const char *prefix_rgb = "output/cutout_output";
      char number_rgb[4] = "000";
      sprintf(number_rgb, "%d", file_index);
      char filename_rgb[strlen(prefix_rgb) + 3 + 4 + 1]; // prefix + number + .ppm + \0
      bzero(filename_rgb, strlen(prefix_rgb) + 3 + 4 + 1);
      strcpy(filename_rgb, prefix_rgb);
      strcpy(filename_rgb + strlen(prefix_rgb), number_rgb);
      strcpy(filename_rgb + strlen(filename_rgb), ".ppm");
      printf("%s\n", filename_rgb);
      writePPM(filename_rgb, buffer_rgb);
  
      file_index += 1;
    }
    destroyPGM(buffer_gray);
    destroyPPM(buffer_rgb);
  } else if (processing_unit == ProcessingUnit::Host) {
    // CPU
    // 1. First step, convert the picture into grayscale
    ProcessingUnitHost::rgb_to_gray(rgb_image, gray_image);
    writePGM("output/gray_image.pgm", gray_image);

    // 2. Second step, smooth the image using a Gaussian blur
    // to remove possible noise in the picture
    for (int i = 0; i < 5; i++) {
      ProcessingUnitHost::gaussian_blur(gray_image->data, gray_image_dim);
    }
    writePGM("output/blurred_image.pgm", gray_image);
    
    // 3. Third step, apply the Sobel-Feldman operator to detect edges of shapes
    ProcessingUnitHost::sobel_feldman(gray_image->data, gradient_image->data, angle_image, gray_image_dim);
    writePGM("output/sf_gradient_output.pgm", gradient_image);
    
    ProcessingUnitHost::generate_edge_color(gradient_image->data, angle_image, edge_color_image->data, rgb_image_dim);
    writePPM("output/edge_color_output.ppm", edge_color_image);
    
    GrayImage *buffer_gray = createPGM(gradient_image->width, gradient_image->height);
    RGBImage *buffer_rgb = createPPM(gradient_image->width, gradient_image->height);
    int file_index = 0;
    for (int i = canny_min; i <= canny_max && canny_sample_offset; i += canny_sample_offset) {
      memcpy(buffer_gray->data, gradient_image->data, sizeof(unsigned char) * gradient_image->width * gradient_image->height);
      ProcessingUnitHost::canny(buffer_gray->data, angle_image, gray_image_dim, i, canny_max);

      // Create the name of the output file
      const char *prefix_gray = "output/canny_output";
      char number_gray[4] = "000";
      sprintf(number_gray, "%d", file_index);
      char filename_gray[strlen(prefix_gray) + 3 + 4 + 1]; // prefix + number + .ppm + \0
      bzero(filename_gray, strlen(prefix_gray) + 3 + 4 + 1);
      strcpy(filename_gray, prefix_gray);
      strcpy(filename_gray + strlen(prefix_gray), number_gray);
      strcpy(filename_gray + strlen(filename_gray), ".ppm");

      //printf("%s\n", filename_gray);
      //writePGM(filename, buffer);
    
      // 4. Last step, cutout the object selected by the user
      memcpy(buffer_rgb->data, rgb_image->data, sizeof(unsigned char) * gradient_image->width * gradient_image->height * 3);
      ProcessingUnitHost::cutout(buffer_rgb->data, buffer_gray->data, gray_image_dim, start_pixel, 0);

      const char *prefix_rgb = "output/cutout_output";
      char number_rgb[4] = "000";
      sprintf(number_rgb, "%d", file_index);
      char filename_rgb[strlen(prefix_rgb) + 3 + 4 + 1]; // prefix + number + .ppm + \0
      bzero(filename_rgb, strlen(prefix_rgb) + 3 + 4 + 1);
      strcpy(filename_rgb, prefix_rgb);
      strcpy(filename_rgb + strlen(prefix_rgb), number_rgb);
      strcpy(filename_rgb + strlen(filename_rgb), ".ppm");
      printf("%s\n", filename_rgb);
      writePPM(filename_rgb, buffer_rgb);
  
      file_index += 1;
    }
    destroyPGM(buffer_gray);
    destroyPPM(buffer_rgb);
  }

  destroyPPM(rgb_image);
  destroyPGM(gray_image);  
  destroyPGM(gradient_image);  
  destroyPPM(edge_color_image);  
  delete [] angle_image; 
}
