#include "hip/hip_runtime.h"
#include <math.h>

#include "sobel_feldman_launcher.hpp"

#include "../../kernel/edge_detection/sobel_feldman_kernel.hpp"
#include "../../core/edge_detection/sobel_feldman_core.hpp"
#include "../../main.hpp"
#include "../../kernel/utils/convolution_kernel.hpp"
#include "../../core/utils/convolution_core.hpp"

const int KERNEL_SIZE = 3;
const float SOBEL_HORIZONTAL_KERNEL[KERNEL_SIZE*KERNEL_SIZE] = { 1, 0,  -1, 
                                                                 2, 0,  -2, 
                                                                 1, 0, -1};
const float SOBEL_VERTICAL_KERNEL[KERNEL_SIZE*KERNEL_SIZE] = { 1,  2,  1,
                                                               0,  0,  0,
                                                               -1, -2, -1}; 
/**
 * Applies the Sobel-Feldman operator over a matrix.
 * The picture should have been smoothed and converted to grayscale prior to being passed over the Sobel-Feldman operator. 
 **/
void ProcessingUnitDevice::sobel_feldman(unsigned char *h_input_matrix, unsigned char *h_gradient_matrix, float *h_angle_matrix, dim3 matrix_dim) {
  dim3 block_dim(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 grid_dim(ceil((float) matrix_dim.x/MATRIX_SIZE_PER_BLOCK), ceil((float) matrix_dim.y/MATRIX_SIZE_PER_BLOCK));

  hipStream_t cuda_streams[2];
  for (int i = 0; i < 2; ++i) {
    hipStreamCreate(&cuda_streams[i]);
  }

  unsigned char *d_input_matrix;
  unsigned char *d_gradient_matrix;
  int *d_horizontal_gradient;
  int *d_vertical_gradient;
  float *d_angle_matrix;
  float *d_horizontal_kernel;
  float *d_vertical_kernel;

  hipMalloc(&d_input_matrix, matrix_dim.x * matrix_dim.y * sizeof(unsigned char));
  hipMalloc(&d_gradient_matrix, matrix_dim.x * matrix_dim.y * sizeof(unsigned char));
  hipMalloc(&d_horizontal_gradient, matrix_dim.x * matrix_dim.y * sizeof(int));
  hipMalloc(&d_vertical_gradient, matrix_dim.x * matrix_dim.y * sizeof(int));
  hipMalloc(&d_angle_matrix, matrix_dim.x * matrix_dim.y * sizeof(float));
  hipMalloc(&d_horizontal_kernel, KERNEL_SIZE*KERNEL_SIZE * sizeof(float));
  hipMalloc(&d_vertical_kernel, KERNEL_SIZE*KERNEL_SIZE * sizeof(float));

  hipMemcpy(d_input_matrix, h_input_matrix, matrix_dim.x * matrix_dim.y * sizeof(unsigned char), hipMemcpyHostToDevice);

  // Horizontal gradient
  hipMemcpyAsync(d_horizontal_kernel, SOBEL_HORIZONTAL_KERNEL, KERNEL_SIZE*KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice, cuda_streams[0]);
  convolution_kernel<<<grid_dim, block_dim, 0, cuda_streams[0]>>>(d_input_matrix, d_horizontal_gradient, matrix_dim, d_horizontal_kernel, 3);

  // Vertical gradient
  hipMemcpyAsync(d_vertical_kernel, SOBEL_VERTICAL_KERNEL, KERNEL_SIZE*KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice, cuda_streams[1]);
  convolution_kernel<<<grid_dim, block_dim, 0, cuda_streams[1]>>>(d_input_matrix, d_vertical_gradient, matrix_dim, d_vertical_kernel, KERNEL_SIZE);

  hipDeviceSynchronize();
  
  // Global gradient
  global_gradient_kernel<<<grid_dim, block_dim, 0, cuda_streams[0]>>>(d_gradient_matrix, d_horizontal_gradient, d_vertical_gradient, matrix_dim); 
  hipMemcpyAsync(h_gradient_matrix, d_gradient_matrix, matrix_dim.x * matrix_dim.y * sizeof(unsigned char), hipMemcpyDeviceToHost, cuda_streams[0]);
 
  // Angle of the gradient
  angle_kernel<<<grid_dim, block_dim, 0, cuda_streams[1]>>>(d_angle_matrix, d_horizontal_gradient, d_vertical_gradient, matrix_dim);
  hipMemcpyAsync(h_angle_matrix, d_angle_matrix, matrix_dim.x * matrix_dim.y * sizeof(float), hipMemcpyDeviceToHost, cuda_streams[1]);

  hipDeviceSynchronize();

  for (int i = 0; i < 2; ++i) {
    hipStreamDestroy(cuda_streams[i]);
  }
  hipFree(d_input_matrix);
  hipFree(d_gradient_matrix);
  hipFree(d_horizontal_gradient);
  hipFree(d_vertical_gradient);
  hipFree(d_angle_matrix);
  hipFree(d_horizontal_kernel);
  hipFree(d_vertical_kernel);
}

void ProcessingUnitHost::sobel_feldman(unsigned char *input_matrix, unsigned char *gradient_matrix, float *angle_matrix, dim3 matrix_dim) {
  int *horizontal_gradient = new int[matrix_dim.x * matrix_dim.y];
  int *vertical_gradient = new int[matrix_dim.x * matrix_dim.y];

  int2 index;

  // Horizontal gradient
  for (index.y = 0; index.y < matrix_dim.y; index.y++) {
    for (index.x = 0; index.x < matrix_dim.x; index.x++) {
      horizontal_gradient[index.y*matrix_dim.x + index.x] = convolution_core(index, input_matrix, matrix_dim, SOBEL_HORIZONTAL_KERNEL, KERNEL_SIZE);
    }
  }

  // Vertical gradient
  for (index.y = 0; index.y < matrix_dim.y; index.y++) {
    for (index.x = 0; index.x < matrix_dim.x; index.x++) {
      vertical_gradient[index.y*matrix_dim.x + index.x] = convolution_core(index, input_matrix, matrix_dim, SOBEL_VERTICAL_KERNEL, KERNEL_SIZE);
    }
  }
  
  // Global gradient
  for (index.y = 0; index.y < matrix_dim.y; index.y++) {
    for (index.x = 0; index.x < matrix_dim.x; index.x++) {
      gradient_matrix[index.y*matrix_dim.x + index.x] = global_gradient_core(index, horizontal_gradient, vertical_gradient, matrix_dim); 
    }
  }
  
  for (index.y = 0; index.y < matrix_dim.y; index.y++) {
    for (index.x = 0; index.x < matrix_dim.x; index.x++) {
      angle_matrix[index.y*matrix_dim.x + index.x] = angle_core(index, horizontal_gradient, vertical_gradient, matrix_dim);
    }
  }
  
  delete [] horizontal_gradient;
  delete [] vertical_gradient;
}



void ProcessingUnitDevice::generate_edge_color(unsigned char *h_gradient_matrix, float *h_angle_matrix, unsigned char *h_output_image, dim3 matrix_dim) {
  dim3 block_dim(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 grid_dim(ceil((float) matrix_dim.x/MATRIX_SIZE_PER_BLOCK), ceil((float) matrix_dim.y/MATRIX_SIZE_PER_BLOCK));

  unsigned char *d_gradient_matrix;
  float *d_angle_matrix;
  unsigned char *d_output_image;

  hipMalloc(&d_gradient_matrix, matrix_dim.x * matrix_dim.y * sizeof(unsigned char));
  hipMalloc(&d_angle_matrix, matrix_dim.x * matrix_dim.y * sizeof(float));
  hipMalloc(&d_output_image, 3 * matrix_dim.x * matrix_dim.y * sizeof(unsigned char));

  hipMemcpy(d_gradient_matrix, h_gradient_matrix, matrix_dim.x * matrix_dim.y * sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_angle_matrix, h_angle_matrix, matrix_dim.x * matrix_dim.y * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_output_image, h_output_image, 3 * matrix_dim.x * matrix_dim.y * sizeof(unsigned char), hipMemcpyHostToDevice);

  edge_color_kernel<<<grid_dim, block_dim>>>(d_gradient_matrix, d_angle_matrix, d_output_image, matrix_dim);

  hipMemcpy(h_output_image, d_output_image, 3 * matrix_dim.x * matrix_dim.y * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(d_gradient_matrix);
  hipFree(d_angle_matrix);
  hipFree(d_output_image);
}

void ProcessingUnitHost::generate_edge_color(unsigned char *gradient_matrix, float *angle_matrix, unsigned char *output_image, dim3 matrix_dim) {
  int2 index;

  for (index.y = 0; index.y < matrix_dim.y; index.y++) {
    for (index.x = 0; index.x < matrix_dim.x; index.x++) {
      edge_color_core(index, gradient_matrix, angle_matrix, output_image, matrix_dim);
    }
  }
}
