#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <iomanip>

#include "main.hpp"
#include "tests.hpp"

int main(int argc, char **argv) {
  //char *filename;
  int2 start_pixel;
  EdgeDetection edge_detection = EdgeDetection::Canny;
  ProcessingUnit processing_unit = ProcessingUnit::Device;
  int canny_min_val = 50;
  int canny_max_val = 100;
  int canny_sample_offset = 0; // Zero: no sample; non-zero value: sample

  if (argc == 2 && strcmp(argv[1], "--help") == 0) {
    print_help();

    return 0;
  } else {
    int i;
    int bad_usage = 0;
    int filename_found = 0;

    for (i = 1; i < argc && !bad_usage; i++) {
      if (strcmp(argv[i], "--start-pixel") == 0) {
        start_pixel.x = atoi(argv[i+1]);
        start_pixel.y = atoi(argv[i+2]);
        i += 2;
      } else if (strcmp(argv[i], "--edge-detection") == 0) {
        if (strcmp(argv[i+1], "sobel") == 0) {
          edge_detection = EdgeDetection::SobelFeldman;
          i += 1;
        } else if (strcmp(argv[i+1], "canny") == 0) {
          edge_detection = EdgeDetection::Canny;
          i += 1;
        } else {
          bad_usage = 1;
        }
      } else if (strcmp(argv[i], "--canny-thresholds") == 0) {
        canny_min_val = atoi(argv[i+1]);
        canny_max_val = atoi(argv[i+2]);
        i += 2;

        if (canny_min_val < 0 || 255 < canny_max_val || canny_max_val < canny_min_val) {
          bad_usage = 1;
        }
      } else if (strcmp(argv[i], "--processing-unit") == 0) {
        if (strcmp(argv[i+1], "host") == 0) {
          processing_unit = ProcessingUnit::Host;
          i += 1;
        } else if (strcmp(argv[i+1], "device") == 0) {
          processing_unit = ProcessingUnit::Device;
          i += 1;
        } else {
          bad_usage = 1;
        }
      } else if (strcmp(argv[i], "--canny-sampling-offset") == 0) {
        canny_sample_offset = atoi(argv[i+1]);
        i += 1;
      } else {
        // This option did not match any possible one
        if (i != argc-1) {
          // Not the filename
          bad_usage = 1;
        } else {
          filename_found = 1;
        }
      }
    }
  
    if (argc == 1 || i == argc && filename_found == 0 || bad_usage) {
      // Filename is missing or bad usage
      print_bad_usage();
      exit(EXIT_FAILURE);
    } else {
      //filename = argv[argc-1];
    }
  }
  
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  
  int file_index = 1;
  std::string prefix("input/frame");

  std::stringstream string_stream;
  std::string number;
  string_stream << std::setw(5) << std::setfill('0') << file_index;
  string_stream >> number;

  std::string filename; // prefix + number + .ppm + \0
  filename.append(prefix);
  filename.append(number);
  filename.append(".ppm");
  std::ifstream file(filename);
  std::cout << filename << std::endl;
  
  while (file.good()) {
    file.close();
    std::cout << filename << std::endl;
    if (edge_detection == EdgeDetection::SobelFeldman) {
      test_sobel_feldman(filename, start_pixel, processing_unit);
    } else if (edge_detection == EdgeDetection::Canny) {
      test_canny(filename, start_pixel, canny_min_val, canny_max_val, canny_sample_offset, processing_unit, file_index);
    }
    if (canny_sample_offset != 0)
      break; // We sample only the first image
    file_index += 1;
    
    std::stringstream string_stream2;
    string_stream2 << std::setw(5) << std::setfill('0') << file_index;
    number.clear();
    string_stream2 >> number;

    filename.clear();
    filename.append(prefix);
    filename.append(number);
    filename.append(".ppm");

    file.open(filename);
  }
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  
  std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::seconds>(end - begin).count() << "s" << std::endl;

  std::cout << " ===" << std::endl;
  hipDeviceSynchronize();
  hipError_t error = hipPeekAtLastError();
  std::cout << "Error: " << hipGetErrorString(error) << std::endl;

  return 0;
}

void print_help() {
  std::cout << "Usage: ./main [OPTION] file" << std::endl;
  std::cout << "\t--start-pixel <x> <y>\t\t\tPixel coordinates where the cutout algorithm should start. (default: 0 0)" << std::endl;

  std::cout << "\t--edge-detection <method>\t\tSpecify the method to use to process edge detection. (default: canny)" << std::endl;
  std::cout << "\t\t\t\t\t\tPermissible methods are 'sobel' and 'canny'." << std::endl;
  std::cout << "\t--canny-thresholds <min> <max>\t\tSpecify the thresholds that have to be used by the Canny edge detector (default: 50 100)" << std::endl;
  std::cout << "\t\t\t\t\t\tPermissible values are integer between 0 and 255." << std::endl;
  std::cout << "\t--processing-unit <processing-unit>\tSpecify where the cutout process has to be executed. (default: device)" << std::endl;
  std::cout << "\t\t\t\t\t\tPermissible processing units are 'host' (CPU) and 'device' (GPU)." << std::endl;
  std::cout << "\t--canny-sampling-offset <offset>\tSpecify that canny should produce multiple outputs, " \
  "starting from the minimum value threshold up to the maximum value" << std::endl;
  std::cout << "\t\t\t\t\t\twith an offset of 'offset' between each sample." << std::endl;

  std::cout << "\t--help\t\t\t\t\tDisplay this help and exit." << std::endl;
}

void print_bad_usage() {
  std::cout << "Usage: ./main [OPTION] file" << std::endl;
  std::cout << "Try './main --help' for more information." << std::endl;
}
