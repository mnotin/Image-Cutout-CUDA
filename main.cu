#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "main.h"
#include "utils.h"
#include "tests.h"

/**
 * Applies discrete convolution over a matrix using a given kernel.
 * This kernel should be called using appropriate number of grids, blocks and threads to match the resolution of the image.
 **/
__global__ void convolution(unsigned char *input_matrix, int *output_matrix, int matrix_width, int matrix_height, float *kernel, int kernel_size) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  int localIdxX = threadIdx.x;
  int localIdxY = threadIdx.y;
  
  int current_matrix_index = globalIdxY*matrix_width + globalIdxX;
  int current_shared_matrix_index = MATRIX_SIZE_PER_BLOCK+2+1+ localIdxY*(MATRIX_SIZE_PER_BLOCK+2) + localIdxX;

  __shared__ unsigned char shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+2)];

  /*
   * x x x x x x MATRIX_SIZE_PER_BLOCK + 2
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x o o o o x
   * x x x x x x
   */
  shared_matrix[current_shared_matrix_index] = input_matrix[current_matrix_index];

  // Handle the borders of each block
  if (localIdxX == 0 && localIdxY == 0) {
    // Fill the edges
    for (int i = 0; i < MATRIX_SIZE_PER_BLOCK+2; i++) {
      // First line
      if (0 < globalIdxY) {
        shared_matrix[i] = input_matrix[(globalIdxY-1)*matrix_width + globalIdxX + i - 1];
      } else {
        shared_matrix[i] = input_matrix[(globalIdxY)*matrix_width + globalIdxX + i - 1];
      }
      
      // Last line
      if (globalIdxY+MATRIX_SIZE_PER_BLOCK < matrix_height) {
        shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+1)+i] =
          input_matrix[(globalIdxY+MATRIX_SIZE_PER_BLOCK)*matrix_width + globalIdxX + i - 1];
      } else {
        shared_matrix[(MATRIX_SIZE_PER_BLOCK+2)*(MATRIX_SIZE_PER_BLOCK+1)+i] =
          input_matrix[(globalIdxY+MATRIX_SIZE_PER_BLOCK-1)*matrix_width + globalIdxX + i - 1];
      }
    }

    for (int i = 0; i < MATRIX_SIZE_PER_BLOCK; i++) {
      // Left side
      if (0 < globalIdxX) {
        shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + i*(MATRIX_SIZE_PER_BLOCK+2)] = input_matrix[(globalIdxY+i)*matrix_width + globalIdxX - 1];
      } else {
        shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + i*(MATRIX_SIZE_PER_BLOCK+2)] = input_matrix[(globalIdxY+i)*matrix_width + globalIdxX];
      }

      // Right side
      if (globalIdxX+MATRIX_SIZE_PER_BLOCK < matrix_width) {
        shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + (i+1)*(MATRIX_SIZE_PER_BLOCK+2) - 1] =
          input_matrix[(globalIdxY+i)*matrix_width + globalIdxX+MATRIX_SIZE_PER_BLOCK];
      } else {
        shared_matrix[MATRIX_SIZE_PER_BLOCK+2 + (i+1)*(MATRIX_SIZE_PER_BLOCK+2) - 1] =
          input_matrix[(globalIdxY+i)*matrix_width + globalIdxX+MATRIX_SIZE_PER_BLOCK-1];
      }
    }
  }
  __syncthreads();

  int convolution_result = 0;

  for (int i = 0; i < kernel_size; i++) {
    for (int j = 0; j < kernel_size; j++) {
      int vertical_offset = ((localIdxY + i) - (int)floor(kernel_size/2.0));
      int horizontal_offset = (localIdxX + j) - (int)floor(kernel_size/2.0);
      int tmp_index = vertical_offset*(MATRIX_SIZE_PER_BLOCK+2) + horizontal_offset;
      
      convolution_result += shared_matrix[MATRIX_SIZE_PER_BLOCK+2+1 + tmp_index] * kernel[i*kernel_size + j];
    }
  }

  if (255 < abs(convolution_result)) {
    convolution_result = convolution_result < 0 ? -255 : 255;
  }
  
  output_matrix[current_matrix_index] = convolution_result;
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("Please provide the name of the file that has to be processed.\n");
    printf("Usage: ./binary filename.ppm start_pixel_x start_pixel_y\n");
    exit(EXIT_FAILURE);
  }

  char *filename = argv[1];
  int start_pixel_x = atoi(argv[2]);
  int start_pixel_y = atoi(argv[3]);

  test_sobel_feldman(filename, start_pixel_x, start_pixel_y);

  printf(" === \n");
  hipDeviceSynchronize();
  hipError_t error = hipPeekAtLastError();
  printf("Error: %s\n", hipGetErrorString(error));

  return 0;
}

