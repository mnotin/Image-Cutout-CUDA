#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "main.h"
#include "edge_detection.h"

#include "tests.h"

/**
 * Applies discrete convolution over a matrix using a given kernel.
 * This kernel should be called using appropriate number of grids, blocks and threads to match the resolution of the image.
 **/
__global__ void convolution(unsigned char *matrix, int matrix_width, int matrix_height, int *kernel, int kernel_size) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);
  int localIdxX = globalIdxX % MATRIX_SIZE_PER_BLOCK;
  int localIdxY = globalIdxY % MATRIX_SIZE_PER_BLOCK;
  
  int current_matrix_index = globalIdxY*matrix_width + globalIdxX;
  int current_shared_matrix_index = localIdxY*MATRIX_SIZE_PER_BLOCK + localIdxX;

  __shared__ unsigned char shared_matrix[MATRIX_SIZE_PER_BLOCK*MATRIX_SIZE_PER_BLOCK];
  shared_matrix[current_shared_matrix_index] = matrix[current_matrix_index];
  __syncthreads();

  int convolution_result = 0;

  if (0 < localIdxX && localIdxX < MATRIX_SIZE_PER_BLOCK-1 && 0 < localIdxY && localIdxY < MATRIX_SIZE_PER_BLOCK-1) {
    for (int i = 0; i < kernel_size; i++) {
      for (int j = 0; j < kernel_size; j++) {
        int vertical_offset = ((localIdxY + i) - (int)floor(kernel_size/2.0));
        int horizontal_offset = (localIdxX + j) - (int)floor(kernel_size/2.0);
        int tmp_index = vertical_offset*MATRIX_SIZE_PER_BLOCK + horizontal_offset;

        convolution_result += shared_matrix[tmp_index] * kernel[i*kernel_size + j];
      }
    }  
  }
  
  matrix[current_matrix_index] = convolution_result;
}

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Please provide the name of the file that has to be processed.\n");
    printf("Usage: ./binary filename.pgm\n");
    exit(EXIT_FAILURE);
  }

  char *filename = argv[1];

  test_sobel_feldman(filename);

  printf(" === \n");
  hipDeviceSynchronize();
  hipError_t error = hipPeekAtLastError();
  printf("Error: %s\n", hipGetErrorString(error));

  return 0;
}

