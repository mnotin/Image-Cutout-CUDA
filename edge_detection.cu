#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "main.h"
#include "edge_detection.h"

/**
 * Applies the Sobel-Feldman operator over a matrix.
 * The picture should have been smoothed and converted to grayscale prior to being passed over the Sobel-Feldman operator. 
 **/
void sobel_feldman(unsigned char *h_matrix, int matrix_width, int matrix_height) {
  const int KERNEL_SIZE = 3;
  int sobel_kernel_horizontal[KERNEL_SIZE*KERNEL_SIZE] = {1, 0,  -1, 
                                                          2, 0,  -2, 
                                                          1, 0, -1};
  int sobel_kernel_vertical[KERNEL_SIZE*KERNEL_SIZE] = { 1,  2,  1,
                                                         0,  0,  0,
                                                        -1, -2, -1};
 
  unsigned char *d_matrix;
  unsigned char *d_horizontal_edges;
  unsigned char *d_vertical_edges;
  int *d_kernel;
  hipMalloc((void **) &d_matrix, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_horizontal_edges, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_vertical_edges, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_kernel, KERNEL_SIZE*KERNEL_SIZE * sizeof(int));

  hipMemcpy(d_horizontal_edges, h_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_vertical_edges, h_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyHostToDevice);

  dim3 threads = dim3(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 blocks = dim3(matrix_width/MATRIX_SIZE_PER_BLOCK, matrix_height/MATRIX_SIZE_PER_BLOCK);
  hipMemcpy(d_kernel, sobel_kernel_horizontal, KERNEL_SIZE*KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);
  printf("Nombre de blocs lancés: %d %d\n", blocks.x, blocks.y);
  convolution<<<blocks, threads>>>(d_horizontal_edges, matrix_width, matrix_height, d_kernel, 3);
  hipMemcpy(d_kernel, sobel_kernel_vertical, KERNEL_SIZE*KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);
  convolution<<<blocks, threads>>>(d_vertical_edges, matrix_width, matrix_height, d_kernel, 3);
  // hipDeviceSynchronize();
  global_gradient<<<blocks, threads>>>(d_matrix, d_horizontal_edges, d_vertical_edges, matrix_width, matrix_height);
 
  hipMemcpy(h_matrix, d_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(d_matrix);
  hipFree(d_horizontal_edges);
  hipFree(d_vertical_edges);
  hipFree(d_kernel);
}

/**
 * Computes the global gradient of an image after being processed by the Sobel-Feldman operator.
 **/
__global__ void global_gradient(unsigned char *matrix, unsigned char *horizontal_edges, unsigned char *vertical_edges, int matrix_width, int matrix_height) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);

  unsigned char g_x = horizontal_edges[globalIdxY*matrix_width + globalIdxX];
  unsigned char g_y = vertical_edges[globalIdxY*matrix_width + globalIdxX];

  matrix[globalIdxY*matrix_width + globalIdxX] = sqrt((double) g_x * g_x + g_y * g_y);
}
