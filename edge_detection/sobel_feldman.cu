#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "sobel_feldman.h"
#include "../main.h"
#include "../utils.h"

/**
 * Applies the Sobel-Feldman operator over a matrix.
 * The picture should have been smoothed and converted to grayscale prior to being passed over the Sobel-Feldman operator. 
 **/
void sobel_feldman(unsigned char *h_matrix, int matrix_width, int matrix_height) {
  const int KERNEL_SIZE = 3;
  float sobel_kernel_horizontal_kernel[KERNEL_SIZE*KERNEL_SIZE] = {1, 0,  -1, 
                                                                   2, 0,  -2, 
                                                                   1, 0, -1};
  float sobel_kernel_vertical_kernel[KERNEL_SIZE*KERNEL_SIZE] = { 1,  2,  1,
                                                                  0,  0,  0,
                                                                 -1, -2, -1}; 
  unsigned char *d_input_matrix;
  unsigned char *d_output_matrix;
  unsigned char *d_horizontal_edges;
  unsigned char *d_vertical_edges;
  float *d_kernel;
  hipMalloc((void **) &d_input_matrix, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_output_matrix, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_horizontal_edges, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_vertical_edges, matrix_width * matrix_height * sizeof(unsigned char));
  hipMalloc((void **) &d_kernel, KERNEL_SIZE*KERNEL_SIZE * sizeof(float));

  hipMemcpy(d_input_matrix, h_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_horizontal_edges, h_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_vertical_edges, h_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyHostToDevice);

  dim3 threads = dim3(MATRIX_SIZE_PER_BLOCK, MATRIX_SIZE_PER_BLOCK);
  dim3 blocks = dim3(matrix_width/MATRIX_SIZE_PER_BLOCK, matrix_height/MATRIX_SIZE_PER_BLOCK);
  hipMemcpy(d_kernel, sobel_kernel_horizontal_kernel, KERNEL_SIZE*KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);
  printf("Nombre de blocs lancés: %d %d\n", blocks.x, blocks.y);
  convolution<<<blocks, threads>>>(d_input_matrix, d_horizontal_edges, matrix_width, matrix_height, d_kernel, 3);
  hipMemcpy(d_kernel, sobel_kernel_vertical_kernel, KERNEL_SIZE*KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);
  convolution<<<blocks, threads>>>(d_input_matrix, d_vertical_edges, matrix_width, matrix_height, d_kernel, 3);
  hipDeviceSynchronize();
  global_gradient<<<blocks, threads>>>(d_output_matrix, d_horizontal_edges, d_vertical_edges, matrix_width, matrix_height);
 
  hipMemcpy(h_matrix, d_output_matrix, matrix_width*matrix_height*sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(d_input_matrix);
  hipFree(d_output_matrix);
  hipFree(d_horizontal_edges);
  hipFree(d_vertical_edges);
  hipFree(d_kernel);
}

/**
 * Computes the global gradient of an image after being processed by the Sobel-Feldman operator.
 **/
__global__ void global_gradient(unsigned char *matrix, unsigned char *horizontal_edges, unsigned char *vertical_edges, int matrix_width, int matrix_height) {
  int globalIdxX = threadIdx.x + (blockIdx.x * blockDim.x);
  int globalIdxY = threadIdx.y + (blockIdx.y * blockDim.y);

  unsigned char g_x = horizontal_edges[globalIdxY*matrix_width + globalIdxX];
  unsigned char g_y = vertical_edges[globalIdxY*matrix_width + globalIdxX];

  matrix[globalIdxY*matrix_width + globalIdxX] = sqrt((double) g_x * g_x + g_y * g_y);
}
